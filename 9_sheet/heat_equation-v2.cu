#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <mpi.h>
#include <math.h>


#define DEFAULT_GRIDSIZE 1024

// Save/ Print only every nth step:
#define PRINTSTEP 10


void init_cells(double* grid, int gridsize);
void print(double* grid, int padded_grid_size, int time);
void save(FILE *f, double* grid, int padded_grid_size, int time);
int find_option( int argc, char **argv, const char *option );
int read_int( int argc, char **argv, const char *option, int default_value );
char *read_string( int argc, char **argv, const char *option, char *default_value );

__global__ void init_matrix(float *A, int rank, int n = 1024) {
  int i = threadIdx.x;

  int i_global = i + rank * 256;

  for(int j=1; j<1024; j++) {
    if(i_global < n/2 && j < n/2 || n/2 >= i_global && n/2 >= j)
      A[i*n+j] = 1;
    else  
      A[i*n+j] = 0;
  }
}

__global__ void jacobi_iteration(float *A_in, float *A_out,
                                int *gt_eps, int rank, float eps = 0.01,
                                int n = 1024) {
  __shared__ int gt_eps_block;
  int i = threadIdx.x;
  if (i == 0) {
    gt_eps_block = 0
  }

  for(int j=1; j<1023; j++) {
    A_out[i*n+j] = (A_in[(i-1)*n+j] + A_in[(i+1)*n+j] \ 
                    + A_in[i*n+j-1] + A_in[i*n+j+1]) / 4.0;
    if(fdimf(A_out[i*n+j], A_in[i*n+j]) > eps)
      gt_eps_block = 1;
  }

  *gt_eps = gt_eps_block;
}

MPI_Status exchange_borders(float *A, int rank, int n = 1024, int processes = 4) {

  MPI_Status status;

  if(rank > 0 && rank < processes-1) {
    MPI_Sendrecv(&A[n], n, MPI_FLOAT, rank-1, 42, 
                  A, n, MPI_FLOAT, rank-1, 42, MPI_COMM_WORLD, &status);
    MPI_Sendrecv(&A[256*n], n, MPI_FLOAT, rank+1, 42, 
                  &A[257*n], n, MPI_FLOAT, rank+1, 42, MPI_COMM_WORLD, &status);
  } else if(rank == 0) {
    MPI_Sendrecv(&A[256*n], n, MPI_FLOAT, 1, 42, &A[257*n], n,
                MPI_FLOAT, 1, 42, MPI_COMM_WORLD, &status);
  } else if(rank == processes-1) {
    MPI_Sendrecv(&A[n], n, MPI_FLOAT, rank-1, 42, A, n,
                MPI_FLOAT, rank-1, 42, MPI_COMM_WORLD, &status);
  }

  return status;
}


int main(int argc, char** argv) {

  int numtasks, rank;
    
  MPI_Init(&argc, &argv);
  MPI_Comm_size(MPI_COMM_WORLD, &numtasks);
  MPI_Comm_rank(MPI_COMM_WORLD, &rank);
  MPI_Status status;
  hipError_t err;
  
  if( find_option( argc, argv, "-h" ) >= 0 )
  {
      printf( "Options:\n" );
      printf( "-h to see this help\n" );
      printf( "-n <int> to set the grid size\n" );
      printf( "-o <filename> to specify the output file name\n" );
      return 1;
  }
  
  int GRIDSIZE = read_int( argc, argv, "-n", DEFAULT_GRIDSIZE );
  // Check gridsize for some basic assumptions
  if(GRIDSIZE != DEFAULT_GRIDSIZE) {
    printf("Only Gridsize of 1024 is allowed!\n");
    return 1;
  }


  char *savename = read_string( argc, argv, "-o", "sample_conduct.txt" );
  FILE *f = savename ? fopen( savename, "w" ) : NULL;
  if( f == NULL )
  {
      printf( "failed to open %s\n", savename );
      return 1;
  }
  

  float *A_block;
  float *A_block_tmp;

  hipMallocManaged(&A_block, sizeof *A_block * 258 * 1024);
  hipMallocManaged(&A_block_tmp, sizeof *A_block_tmp * 258 * 1024);
  init_matrix<<<1, 256>>>(&A_block[1024], rank);

  err = hipGetLastError();    
  if (err != hipSuccess) 
    fprintf(stderr, "Error: %s\n", hipGetErrorString(err));
  hipDeviceSynchronize();

  int *gt_all_eps;
  hipMallocManaged(&gt_all_eps, sizeof *gt_all_eps * 4);
  bool gt_eps_global = true;


  while(gt_eps_global) {
    exchange_borders(A_block, rank);
    if(rank == 0)
      jacobi_iteration<<<1, 255>>>(&A_block[2*1024], &A_block_tmp[2*1024],
                                   &gt_all_eps[rank], rank);
    else if(rank == 3)
      jacobi_iteration<<<1, 255>>>(&A_block[1*1024], &A_block_tmp[1*1024],
        &gt_all_eps[rank], rank);
    else 
      jacobi_iteration<<<1, 256>>>(&A_block[1*1024], &A_block_tmp[1*1024],
        &gt_all_eps[rank], rank);

    err = hipGetLastError();    
    if (err != hipSuccess) 
      fprintf(stderr, "Error: %s\n", hipGetErrorString(err));
    hipDeviceSynchronize();
    
    MPI_Gather(MPI_IN_PLACE, 1, MPI_INT, gt_all_eps, 1, MPI_INT, 0, MPI_COMM_WORLD);
    for(int i=0; i<4; i++)
      gt_all_eps |= gt_all_eps[i];
  }

  MPI_Finalize();
  return 0;
}



void print(double* grid, int padded_grid_size, int time) {
  printf("\n\n\n");
  int i,j;
  
  // we don't want to print the border!
  for(i=1;i<padded_grid_size-1;i++) {
    for(j=1;j<padded_grid_size-1;j++) {
      printf("%.2f ",grid[i*padded_grid_size + j]);
    }
    printf("\n");
  }
}

void save( FILE *f, double* grid, int padded_grid_size,int TIMESTEPS)
{
    int i,j;
    
    static int first = 1;
    if( first )
    {
    fprintf( f, "# %d %d\n", TIMESTEPS, padded_grid_size-2 );
        first = 0;
    }

    for(i = 1; i < padded_grid_size-1; i++ ) {
      for(j=1; j < padded_grid_size-1; j++) {
        fprintf( f, "%.g ", grid[i* padded_grid_size + j] );
      }
      fprintf(f,"\n");
    }

}


//
//  command line option processing
//
int find_option( int argc, char **argv, const char *option )
{
  int i;
  for( i = 1; i < argc; i++ )
    if( strcmp( argv[i], option ) == 0 )
      return i;
  return -1;
}

int read_int( int argc, char **argv, const char *option, int default_value )
{
  int iplace = find_option( argc, argv, option );
  if( iplace >= 0 && iplace < argc-1 )
    return atoi( argv[iplace+1] );
  return default_value;
}

char *read_string( int argc, char **argv, const char *option, char *default_value )
{
  int iplace = find_option( argc, argv, option );
  if( iplace >= 0 && iplace < argc-1 )
    return argv[iplace+1];
  return default_value;
}
