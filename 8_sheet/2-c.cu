#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>

bool is_power_of_two(unsigned int x)
{
    // https://stackoverflow.com/questions/600293/how-to-check-if-a-number-is-a-power-of-2/600306#600306
    return (x != 0) && ((x & (x - 1)) == 0);
}


template <unsigned int blockSize>
__global__ void reduce(float *g_in, double *g_out, unsigned int n)
{
    extern __shared__ double s_data[];

    unsigned int tid = threadIdx.x;
    unsigned int i = blockIdx.x*(blockSize * 2) + tid;
    unsigned int gridSize = blockSize * 2 * gridDim.x;
    s_data[tid] = 0;
    
    while(i < n) {
        s_data[tid] += g_in[i] + g_in[i+blockSize];
        i += gridSize;
    }
    __syncthreads();

    if(blockSize >= 512) {
        if(tid < 256)
            s_data[tid] += s_data[tid + 256];
        __syncthreads();
    }
    if(blockSize >= 256) {
        if(tid < 128)
            s_data[tid] += s_data[tid + 128];
        __syncthreads();
    }
    if(blockSize >= 128) {
        if(tid < 64)
            s_data[tid] += s_data[tid + 64];
        __syncthreads();
    }

    if(tid < 32) {
        if(blockSize >= 64) s_data[tid] += s_data[tid + 32];
        if(blockSize >= 32) s_data[tid] += s_data[tid + 16];
        if(blockSize >= 16) s_data[tid] += s_data[tid + 8];
        if(blockSize >= 8) s_data[tid] += s_data[tid + 4];
        if(blockSize >= 4) s_data[tid] += s_data[tid + 2];
        if(blockSize >= 2) s_data[tid] += s_data[tid + 1];
    }
    if(tid == 0) {
        printf("Writing %f\n", s_data[0]);
        g_out[blockIdx.x] = s_data[0];
    }
}


int main(int argc, char const *argv[])
{
    if(argc <= 1) {
        printf("Please call the program with an integer that is a"
                " power of two.");
        return 1;
    }
    unsigned int N = atoi(argv[1]);
    if(!is_power_of_two(N)) {
        printf("Please call the program with an integer that is a"
                " power of two.");
        return 1;
    }

    float *in;
    double *out; 
    hipMallocManaged(&in, sizeof *in * N);
    
    // initialize array on host with ones
    for(int i=0; i<N; i++) {
        in[i] = 1;
    }


    int threadCountGrid = ceil((double)N / log2(N));
    int dimBlock = 512;
    printf("Using %d threads in total\n", max(threadCountGrid, dimBlock);
    printf("Divided on blocks of size %d\n", dimBlock);    
    int dimGrid = ceil((double)threadCountGrid / 512);
    printf("With %d blocks\n", dimGrid);    
    size_t smemSize = sizeof *out * dimBlock;
    hipMallocManaged(&out, sizeof *out * dimGrid);

    switch(dimBlock) {
        case 512:
            reduce<512><<<dimGrid, dimBlock, smemSize>>>(in, out, N);
            break;
        case 256:
            reduce<256><<<dimGrid, dimBlock, smemSize>>>(in, out, N);
            break;
        case 128:
            reduce<128><<<dimGrid, dimBlock, smemSize>>>(in, out, N);
            break;
        case 64:
            reduce<64><<<dimGrid, dimBlock, smemSize>>>(in, out, N);
            break;
        case 32:
            reduce<32><<<dimGrid, dimBlock, smemSize>>>(in, out, N);
            break;
        case 16:
            reduce<16><<<dimGrid, dimBlock, smemSize>>>(in, out, N);
            break;
        case 8:
            reduce<8><<<dimGrid, dimBlock, smemSize>>>(in, out, N);
            break;
        case 4:
            reduce<4><<<dimGrid, dimBlock, smemSize>>>(in, out, N);
            break;
        case 2:
            reduce<2><<<dimGrid, dimBlock, smemSize>>>(in, out, N);
            break;
        case 1:
            reduce<1><<<dimGrid, dimBlock, smemSize>>>(in, out, N);
            break;
    }

    hipDeviceSynchronize();    

    for(int i=0; i<dimGrid; i++) {
        printf("%f,", out[i]);
    }
    printf("\n");    

    hipFree(in);
    hipFree(out);

    return 0;


}
