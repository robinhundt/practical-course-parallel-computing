
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

int is_power_of_two(int x)
{
    // https://stackoverflow.com/questions/600293/how-to-check-if-a-number-is-a-power-of-2/600306#600306
    return (x != 0) && ((x & (x - 1)) == 0);
}

__global__ void  sum_arr(float *arr, int n, double *res) { 
    for(int i=0; i<n; i++)
        *res += arr[i];
}

int main(int argc, char const *argv[])
{
    const int N = argc-1;    
    if(N == 0 || !is_power_of_two(N)) {
        printf("Please call the program with the 2^N array elements "
                "to be summed as arguments.\n");
        return 1;
    }

    float *arr;
    double *sum; 
    hipMallocManaged(&arr, sizeof *arr * N);
    hipMallocManaged(&sum, sizeof *sum);
    *sum = 0;
    
    // initialize array on host with program arguments
    for(int i=0; i<N; i++) {
        arr[i] = atof(argv[i+1]);
    }
    sum_arr<<<1, 1>>>(arr, N, sum);
    hipDeviceSynchronize();

    hipFree(arr);

    printf("Result is: %f\n", *sum);
    return 0;
}
