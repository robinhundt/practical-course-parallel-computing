
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>
#include <cmath>
#include <cstring>
#include <algorithm>


using namespace std;

__global__ void initMatrix(float *A, int n = 512) {
    uint i = (blockIdx.y * blockDim.y) + threadIdx.y;
    uint j = (blockIdx.x * blockDim.x) + threadIdx.x;
    // printf("j: %d, i: %d\n", i, j);
    if(i == 0 || i == n-1 || j == 0 || j == n-1)
        A[i*n+j] = cosf((4.0 * M_PI * i) / (n - 1.0)) \
                    * cosf((4.0 * M_PI * j) / (n - 1.0));
    else
        A[i*n+j] = 0;
}

 __global__ void jacobiIteration(float *A, int n = 512, int iterations = 100) {
    float A_nghbrs[4];
    uint i = (blockIdx.y * blockDim.y) + threadIdx.y;
    uint j = (blockIdx.x * blockDim.x) + threadIdx.x;
    for(int k=0; k<iterations; k++) {
        __syncthreads();
        A_nghbrs[0] = A[(i-1)*n+j];
        A_nghbrs[1] = A[(i+1)*n+j];
        A_nghbrs[2] = A[i*n+j-1];
        A_nghbrs[3] = A[i*n+j+1];
        __syncthreads();
        A[i*n+j] = (A_nghbrs[0] + A_nghbrs[1] + A_nghbrs[2] + A_nghbrs[3]) / 4.0;
    }
}

void printMatrix(float *A, int n = 512) {
    for(int i=0; i<n; i++) {
        for(int  j=0; j<n-1; j++) {
            printf("%f;", A[i*n+j]);
        }
        printf("%f\n", A[i*n+n-1]);        
    }
}

int main(int argc, char *argv[]) {
    unsigned int N;
    int iterations = 36;
    if (argc > 1)
        N = atoi(argv[1]);
    else
        N = 512;
    fprintf(stderr, "Using grid size %dx%d\n", N, N);                
    float *A;
    hipMallocManaged(&A, sizeof *A *N*N);
    dim3 nt(8,8);
    dim3 nb(N/nt.x, N/nt.y);
    initMatrix<<<nb, nt>>>(A, N);
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) 
        printf("Error: %s\n", hipGetErrorString(err));
    hipDeviceSynchronize();        
    printMatrix(A, N);
    jacobiIteration<<<nt, nb>>>(A, N, iterations);
    printMatrix(A, N);
    fprintf(stderr, "iterations: %d\n", iterations);    
    hipFree(A);
    return 0;
}